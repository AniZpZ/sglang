#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Adapted from https://github.com/IST-DASLab/marlin
 */

 #ifndef MARLIN_W4A8_NAMESPACE_NAME
 #define MARLIN_W4A8_NAMESPACE_NAME marlin_moe_w4a8
#endif

#include "kernel.h"
#include "core/registration.h"

#define STATIC_ASSERT_SCALAR_TYPE_VALID(scalar_t)               \
 static_assert(std::is_same<scalar_t, half>::value ||          \
                   std::is_same<scalar_t, hip_bfloat16>::value, \
               "only float16 and bfloat16 is supported");

namespace MARLIN_W4A8_NAMESPACE_NAME {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800


}  // namespace marlin

torch::Tensor moe_w4a8_marlin_gemm(
   torch::Tensor& a, std::optional<torch::Tensor> const& c_or_none,
   torch::Tensor& b_q_weight, torch::Tensor& b_scales,
   std::optional<torch::Tensor> const& b_zeros_or_none,
   std::optional<torch::Tensor> const& g_idx_or_none,
   std::optional<torch::Tensor> const& perm_or_none, torch::Tensor& workspace,
   torch::Tensor& sorted_token_ids, torch::Tensor& expert_ids,
   torch::Tensor& num_tokens_past_padded, torch::Tensor& topk_weights,
   int64_t moe_block_size, int64_t top_k, bool mul_topk_weights, bool is_ep,
   vllm::ScalarTypeId const& b_q_type_id, int64_t size_m, int64_t size_n,
   int64_t size_k, bool is_k_full, bool use_atomic_add, bool use_fp32_reduce,
   bool is_zp_float) {
 TORCH_CHECK_NOT_IMPLEMENTED(false,
                             "marlin_gemm(..) requires CUDA_ARCH >= 8.0");
 return torch::empty({1, 1});
}

#else


typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},  // Default
    {128, 64, 128},   // Reduce N 2X, same K
    {64, 256, 256},   // Reduce K 2X, increase N 2X
    {64, 128, 128},   // Reduce K 2X, same N
};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},   // Default
    {128, 128, 256},  // Reduce N 2X, increase K 2X
    {64, 128, 128},   // Reduce N 2X, same K
    {128, 64, 128},   // Reduce N 4X, increase K 2X
};

int get_scales_cache_size(thread_config_t const& th_config, int prob_m,
                          int prob_n, int prob_k, int group_size) {

  int tb_n = th_config.thread_n;
  int tb_k = th_config.thread_k;

  // Get max scale groups per thread-block
  int tb_groups;
  if (group_size == -1) {
    tb_groups = 1;
  } else if (group_size == 0) {
    tb_groups = div_ceil(tb_k, 32);  // Worst case is 32 group size
  } else {
    tb_groups = div_ceil(tb_k, group_size);
  }

  int tb_scales = tb_groups * tb_n * 2;

  return tb_scales * pipe_stages;
}

bool is_valid_cache_size(thread_config_t const& th_config, int moe_block_size,
                         int prob_m, int prob_n, int prob_k,
                         int scales_cache_size, int max_shared_mem) {
  int pack_factor = 8;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;

  int b_size = (tb_k * tb_n / pack_factor) * 4;

  // Get A size
  int tb_max_m = moe_block_size;
  int a_size = (tb_max_m * tb_k) * 2;

  float pipe_size = (a_size + b_size) * pipe_stages;

  float reduce_size = max(th_config.num_threads * 32 * 4,
                          (tb_n / 64) * 32 * (tb_max_m / 16) * 4 * 2 * 4 * 2);

  TORCH_CHECK(max_shared_mem / 2 > scales_cache_size);  // Sanity

  return pipe_size + reduce_size < 0.95f * (max_shared_mem - scales_cache_size);
}

bool is_valid_config(thread_config_t const& th_config, int moe_block_size, int prob_m, int prob_n,
                     int prob_k, int group_size, int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 ||
      th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // thread_k can be only 128 or 64 (because it must be less than groupsize
  // which is 128)
  if (th_config.thread_k != 128 && th_config.thread_k != 64) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  //  Determine cache for scales
  int scales_cache_size =
      get_scales_cache_size(th_config, prob_m, prob_n, prob_k,
                            group_size);

  // Check that pipeline fits into cache
  if (!is_valid_cache_size(th_config, moe_block_size, prob_m, prob_n, prob_k,
                           scales_cache_size, max_shared_mem)) {
    return false;
  }

  return true;
}

thread_config_t determine_thread_config(int prob_m, int prob_n, int prob_k
                                        int moe_block_size,int group_size,
                                        int max_shared_mem) {
  if (moe_block_size <= 16) {
    for (auto th_config : small_batch_thread_configs) {
      if (is_valid_config(th_config, moe_block_size, prob_m, prob_n, prob_k,
                          group_size, max_shared_mem)) {
        return th_config;
      }
    }

  } else {
    for (auto th_config : large_batch_thread_configs) {
      if (is_valid_config(th_config, moe_block_size, prob_m, prob_n, prob_k,
                          group_size, max_shared_mem)) {
        return th_config;
      }
    }
  }

  return thread_config_t{-1, -1, -1};
}


#define __CALL_IF(THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,               \
    GROUP_BLOCKS, NUM_THREADS)                                       \
else if (thread_m_blocks == THREAD_M_BLOCKS &&                                   \
thread_n_blocks == THREAD_N_BLOCKS &&                                   \
thread_k_blocks == THREAD_K_BLOCKS &&                                   \
group_blocks == GROUP_BLOCKS && num_threads == NUM_THREADS) {           \
hipFuncSetAttribute(reinterpret_cast<const void*>(Marlin<NUM_THREADS), THREAD_M_BLOCKS, THREAD_N_BLOCKS,     \
                  THREAD_K_BLOCKS, pipe_stages, GROUP_BLOCKS>,            \
           hipFuncAttributeMaxDynamicSharedMemorySize,              \
           max_shared_mem);                                          \
Marlin<NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,         \
pipe_stages, GROUP_BLOCKS>                                                   \
<<<blocks, NUM_THREADS, max_shared_mem, stream>>>(                         \
A_ptr, B_ptr, C_ptr, D_ptr, s1_ptr, s2_ptr, s3_ptr,                    \
sorted_token_ids_ptr, expert_ids_ptr,                                  \
num_tokens_past_padded_ptr, topk_weights_ptr, top_k,                   \
mul_topk_weights, is_ep, num_groups,                                   \
prob_m, prob_n, prob_k, locks);                                        \
}

#define CALL_IF(N_BLOCKS, K_BLOCKS, NUM_THREADS)    \
__CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
__CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
__CALL_IF(1, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
__CALL_IF(1, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
__CALL_IF(2, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
__CALL_IF(2, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
__CALL_IF(3, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
__CALL_IF(3, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)  \
__CALL_IF(4, N_BLOCKS, K_BLOCKS, -1, NUM_THREADS) \
__CALL_IF(4, N_BLOCKS, K_BLOCKS, 8, NUM_THREADS)

const int ERR_PROB_SHAPE = 1;
const int ERR_KERN_SHAPE = 2;

void marlin_w4a8_mm(
    const void* A,
    const void* B,
          void* C, // int32 reduce buffer
          void* D, // half
    const void* s1,
    const void* s2,
    const void* s3,
    void* sorted_token_ids,  //moe start
    void* expert_ids,
    void* num_tokens_past_padded, 
    void* topk_weights,
    int moe_block_size, 
    int top_k, 
    bool mul_topk_weights, 
    bool is_ep,             //moe end
    int prob_m,
    int prob_n,
    int prob_k,
    void* workspace,
    int groupsize = -1,
    bool is_k_full, //optional
    int dev = 0,
    hipStream_t stream = 0,
    int thread_k = -1,
    int thread_n = -1,
    int sms = -1,
    int max_par = 16
  ) {
    int thread_m_blocks = moe_block_size / 16;
  
    int tot_m = prob_m;
    int tot_m_blocks = ceildiv(tot_m, 16);
    
    // hongbo: whether need it
    int pad = 16 * tot_m_blocks - tot_m;
  
    if (sms == -1)
      hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, dev);
    int max_shared_mem = 0;
    hipDeviceGetAttribute(&max_shared_mem,
                           hipDeviceAttributeSharedMemPerBlockOptin, dev);
  
    // Set thread config
    thread_config_t th_config;
    if (thread_k != -1 && thread_n != -1) {
      // User-defined config
      th_config = thread_config_t{thread_k, thread_n, USER_THREADS};
    } else {
      // Auto config
      th_config = determine_thread_config(prob_m, prob_n, prob_k, moe_block_size,
                                          group_size, max_shared_mem);
    }
    // int group_blocks = (groupsize == -1) ? -1 : groupsize / 16;
    int group_blocks = 0;
    if (group_size == -1) {
        group_blocks = -1;
    } else {
      group_blocks = group_size / 16;
      TORCH_CHECK(prob_k % group_blocks == 0, "prob_k = ", prob_k,
                    " is not divisible by group_blocks = ", group_blocks);
    }
    
    if (!is_valid_config(th_config, prob_m, prob_n, prob_k) || (group_blocks != -1 && prob_k % group_blocks != 0))
      return ERR_PROB_SHAPE;
    TORCH_CHECK(!is_valid_config(th_config, prob_m, prob_n, prob_k) || (group_blocks != -1 && prob_k % group_blocks != 0),
                "Invalid thread config: moe_block_size = ", 0,
                ", thread_k = ", th_config.thread_k,
                ", thread_n = ", th_config.thread_n,
                ", num_threads = ", th_config.num_threads, " for MKN = [",
                prob_m, ", ", prob_k, ", ", prob_n, "] and num_bits = ", 4,
                ", group_size = ", group_size,);
    
    int num_threads = th_config.num_threads;
    thread_k = th_config.thread_k;
    thread_n = th_config.thread_n;
    int thread_k_blocks = thread_k / 16;
    int thread_n_blocks = thread_n / 16;
    int blocks = sms;
  
    if (groupsize == -1)
      assert(s3 == nullptr);
    if (prob_m == 0 || prob_n == 0 || prob_k == 0)
      return 0;
    TORCH_CHECK(prob_n % thread_n == 0, "prob_n = ", prob_n,
                " is not divisible by thread_n = ", thread_n);
    TORCH_CHECK(prob_k % thread_k == 0, "prob_k = ", prob_k,
                " is not divisible by thread_k = ", thread_k);
  
    const int4* A_ptr = (const int4*) A;
    const int4* B_ptr = (const int4*) B;
    int4* C_ptr = (int4*) C;
    int4* D_ptr = (int4*) D;
    const float* s1_ptr = (const float*) s1;
    const int4* s2_ptr = (const int4*) s2;
    const int4* s3_ptr = (const int4*) s3;
    // moe param
    const int32_t* sorted_token_ids_ptr = (const int32_t*)sorted_token_ids;
    const int32_t* expert_ids_ptr = (const int32_t*)expert_ids;
    const int32_t* num_tokens_past_padded_ptr =
        (const int32_t*)num_tokens_past_padded;
    const float* topk_weights_ptr = (const float*)topk_weights;
    // moe param
    int* locks = (int*) workspace;
  
    if (false) {}
      CALL_IF(8, 8, 256)
      CALL_IF(16, 4, 256)
      CALL_IF(8, 4, 128)
      CALL_IF(4, 8, 128)
    else {
      TORCH_CHECK(false, "Unsupported shapes: MNK = [", prob_m, ", ", prob_n,
                  ", ", prob_k, "]",  ", group_size = ", group_size,
                  ", thread_m_blocks = ", thread_m_blocks,
                  ", thread_n_blocks = ", thread_n_blocks,
                  ", thread_k_blocks = ", thread_k_blocks);
    }
  
  }  

}  // namespace MARLIN_W4A8_NAMESPACE_NAME

torch::Tensor moe_w4a8_marlin_gemm(
    torch::Tensor& a, torch::Tensor& a_scale,
    std::optional<torch::Tensor> const& c_or_none,
    torch::Tensor& b_q_weight, torch::Tensor& b_scales,
    std::optional<torch::Tensor> const& b_zeros_or_none,
    std::optional<torch::Tensor> const& g_idx_or_none,
    std::optional<torch::Tensor> const& perm_or_none, torch::Tensor& workspace,
    torch::Tensor& sorted_token_ids, torch::Tensor& expert_ids,
    torch::Tensor& num_tokens_past_padded, torch::Tensor& topk_weights,
    int64_t moe_block_size, int64_t top_k, bool mul_topk_weights, bool is_ep,
    sglang::ScalarTypeId const& b_q_type_id, int64_t size_m, int64_t size_n,
    int64_t size_k, bool is_k_full, bool use_atomic_add, bool use_fp32_reduce,
    bool is_zp_float) {
  sglang::ScalarType const b_q_type = sglang::ScalarType::from_id(b_q_type_id);
  int pack_factor = 32 / b_q_type.size_bits();

  // Verify A
  TORCH_CHECK(a.size(0) == size_m, "Shape mismatch: a.size(0) = ", a.size(0),
              ", size_m = ", size_m);
  TORCH_CHECK(a.size(1) == size_k, "Shape mismatch: a.size(1) = ", a.size(1),
              ", size_k = ", size_k);

  // Verify B
  TORCH_CHECK(
      size_k % MARLIN_NAMESPACE_NAME::tile_size == 0, "size_k = ", size_k,
      " is not divisible by tile_size = ", MARLIN_NAMESPACE_NAME::tile_size);
  TORCH_CHECK((size_k / MARLIN_NAMESPACE_NAME::tile_size) == b_q_weight.size(1),
              "Shape mismatch: b_q_weight.size(1) = ", b_q_weight.size(1),
              ", size_k = ", size_k,
              ", tile_size = ", MARLIN_NAMESPACE_NAME::tile_size);
  TORCH_CHECK(
      b_q_weight.size(2) % MARLIN_NAMESPACE_NAME::tile_size == 0,
      "b_q_weight.size(2) = ", b_q_weight.size(2),
      " is not divisible by tile_size = ", MARLIN_NAMESPACE_NAME::tile_size);
  int actual_size_n =
      (b_q_weight.size(2) / MARLIN_NAMESPACE_NAME::tile_size) * pack_factor;
  TORCH_CHECK(size_n == actual_size_n, "size_n = ", size_n,
              ", actual_size_n = ", actual_size_n);

  // Verify device and strides
  TORCH_CHECK(a.device().is_cuda(), "A is not on GPU");
  TORCH_CHECK(a.is_contiguous(), "A is not contiguous");

  TORCH_CHECK(b_q_weight.device().is_cuda(), "b_q_weight is not on GPU");
  TORCH_CHECK(b_q_weight.is_contiguous(), "b_q_weight is not contiguous");

  TORCH_CHECK(b_scales.device().is_cuda(), "b_scales is not on GPU");
  TORCH_CHECK(b_scales.is_contiguous(), "b_scales is not contiguous");

  // thread_k: `k` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_k = -1;
  // thread_n: `n` size of a thread_tile in `weights` (can usually be left as
  // auto -1)
  int thread_n = -1;
  // sms: number of SMs to use for the kernel
  int sms = -1;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, a.get_device());

  // Alloc buffers
  const at::cuda::OptionalCUDAGuard device_guard(device_of(a));
  auto options = torch::TensorOptions().dtype(a.dtype()).device(a.device());
  torch::Tensor c;
  if (c_or_none.has_value()) {
    c = c_or_none.value();
    TORCH_CHECK(c.device().is_cuda(), "c is not on GPU");
    TORCH_CHECK(c.is_contiguous(), "c is not contiguous");
    TORCH_CHECK(c.size(0) == size_m * top_k,
                "Shape mismatch: c.size(0) = ", c.size(0),
                ", size_m * topk = ", size_m * top_k);
    TORCH_CHECK(c.size(1) == size_n, "Shape mismatch: c.size(1) = ", c.size(1),
                ", size_n = ", size_n);
  } else {
    c = torch::empty({size_m * top_k, size_n}, options);
  }

  // Alloc C tmp buffer that is going to be used for the global reduce
  torch::Tensor c_tmp;
  auto options_fp32 =
      torch::TensorOptions().dtype(at::kFloat).device(a.device());
  if (use_fp32_reduce && !use_atomic_add) {
    const long max_c_tmp_size =
        min(((long)size_n * sorted_token_ids.size(0)),
            (long)(sms * moe_block_size * MARLIN_NAMESPACE_NAME::max_thread_n));
    c_tmp = torch::empty({max_c_tmp_size}, options_fp32);
  } else {
    c_tmp = torch::empty({0}, options_fp32);
  }

  // Detect groupsize and act_order
  int num_groups = -1;
  int group_size = -1;

  int rank = b_scales.sizes().size();
  TORCH_CHECK(rank == 3, "b_scales rank = ", rank, " is not 3");
  TORCH_CHECK(b_scales.size(2) == size_n, "b_scales dim 2 = ", b_scales.size(2),
              " is not size_n = ", size_n);
  num_groups = b_scales.size(1);

  torch::Tensor g_idx, perm, a_tmp;
  ;
  if (g_idx_or_none.has_value() && perm_or_none.has_value()) {
    g_idx = g_idx_or_none.value();
    perm = perm_or_none.value();

    TORCH_CHECK(g_idx.device().is_cuda(), "g_idx is not on GPU");
    TORCH_CHECK(g_idx.is_contiguous(), "g_idx is not contiguous");
    TORCH_CHECK(perm.device().is_cuda(), "perm is not on GPU");
    TORCH_CHECK(perm.is_contiguous(), "perm is not contiguous");

    // Verify g_idx and perm
    TORCH_CHECK((g_idx.size(-1) == 0 && perm.size(-1) == 0) ||
                    (g_idx.size(-1) == size_k && perm.size(-1) == size_k),
                "Unexpected g_idx.size(-1) = ", g_idx.size(-1),
                " and perm.size(-1) = ", perm.size(-1),
                ", where size_k = ", size_k);
  } else {
    g_idx = torch::empty({0}, options);
    perm = torch::empty({0}, options);
    a_tmp = torch::empty({0}, options);
  }
  bool has_act_order = g_idx.size(-1) > 0 && perm.size(-1) > 0;

  if (has_act_order) {
    a_tmp = torch::empty({size_m * top_k, size_k}, options);
    if (is_k_full) {
      TORCH_CHECK(num_groups > 1, "For act_order, num_groups must be > 1");
      TORCH_CHECK(size_k % num_groups == 0, "size_k = ", size_k,
                  ", is not divisible by num_groups = ", num_groups);
      group_size = size_k / num_groups;
    } else {
      group_size = 0;
    }

  } else {
    a_tmp = torch::empty({0}, options);
    if (num_groups > 1) {
      TORCH_CHECK(
          size_k % num_groups == 0, "size_k = ", size_k,
          ", is not divisible by b_scales.size(1) = ", b_scales.size(1));
      group_size = size_k / num_groups;
    } else {
      group_size = -1;
    }
  }

  // Verify workspace size
  TORCH_CHECK(size_n % MARLIN_NAMESPACE_NAME::min_thread_n == 0,
              "size_n = ", size_n, ", is not divisible by min_thread_n = ",
              MARLIN_NAMESPACE_NAME::min_thread_n);

  int max_n_tiles = size_n / MARLIN_NAMESPACE_NAME::min_thread_n;
  int min_workspace_size =
      min(max_n_tiles * (int)(sorted_token_ids.size(0) / moe_block_size), sms);
  TORCH_CHECK(workspace.numel() >= min_workspace_size,
              "workspace.numel = ", workspace.numel(),
              " is below min_workspace_size = ", min_workspace_size);

  int dev = a.get_device();
  marlin_qqq_cuda(
      a.data_ptr(), b_q_weight.data_ptr(), c.data_ptr(), d.data_ptr(),
      s_tok.data_ptr(), s_ch.data_ptr(), s_group.data_ptr(), size_m, size_n,
      size_k, workspace.data_ptr(), groupsize, dev,
      at::cuda::getCurrentCUDAStream(dev), thread_k, thread_n, sms, max_par);

  return c;
}

#endif